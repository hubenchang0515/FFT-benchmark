#include "hip/hip_runtime.h"
#include "config.h"
#include <cstdio>
#include <cstdlib>
#include <cuda/std/cmath>
#include <cuda/std/complex>

using Complex = cuda::std::complex<double>;

__device__ 
Complex omega(int k, int n)
{
    double angle = 2.0*pi*k/n;
    return Complex{cuda::std::cos(angle), -cuda::std::sin(angle)};
}

__global__
void polynomialItems(const Complex* K, size_t n, Complex* items, int forward=1)
{
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (bid >= n || tid >= n)
        return;

    items[i] = K[tid] * cuda::std::pow(omega(forward * bid, n), tid);
}

__global__
void sumItems(const Complex* items, size_t n, Complex* values)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    for (size_t j = 0; j < n; j++)
    {
        values[i] += items[i * n + j];
    }
}

__device__
Complex polynomialValue(const Complex* K, size_t n, const Complex& w)
{
    Complex v = 0;
    for (size_t i = 0; i < n; i++)
    {
        v += K[i] * cuda::std::pow(w, i);
    }

    return v;
}

__global__
void transformForDft(const Complex* in, size_t n, Complex* out, int forward=1)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    out[i] = polynomialValue(in, n, omega(forward * i, n));
}

__global__
void div(Complex* data, size_t n, double v)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    data[i] /= v;
}

// 显存开销和并法规模较大的方案
void dft1(const Complex* in, size_t n, Complex* out, int forward=1)
{
    // 分配显存
    Complex* devIn;
    Complex* devOut;
    Complex* devItems;
    hipMalloc(&devIn, n * sizeof(Complex));
    hipMalloc(&devOut, n * sizeof(Complex));
    hipMalloc(&devItems, n * n * sizeof(Complex));

    // 将内存数据拷贝到显存
    hipMemcpy(devIn, in, n * sizeof(Complex), hipMemcpyHostToDevice);

    // 计算
    polynomialItems<<<n, n>>>(devIn, n, devItems, forward);
    sumItems<<<1, n>>>(devItems, n, devOut);

    if (forward < 0)
    {
        div<<<1, n>>>(devOut, n, n);
    }

    // 将显存数据拷贝到内存
    hipMemcpy(out, devOut, n * sizeof(Complex), hipMemcpyDeviceToHost);

    // 释放显存
    hipFree(devItems);
    hipFree(devOut);
    hipFree(devIn);
}

// 显存开销和并法规模较小的方案
void dft2(const Complex* in, size_t n, Complex* out, int forward=1)
{
    // 分配显存
    Complex* devIn;
    Complex* devOut;
    hipMalloc(&devIn, n * sizeof(Complex));
    hipMalloc(&devOut, n * sizeof(Complex));

    // 将内存数据拷贝到显存
    hipMemcpy(devIn, in, n * sizeof(Complex), hipMemcpyHostToDevice);

    transformForDft<<<(n+1023)/1024, 1024>>>(devIn, n, devOut, forward);

    if (forward < 0)
    {
        div<<<(n+1023)/1024, 1024>>>(devOut, n, n);
    }

    // 将显存数据拷贝到内存
    hipMemcpy(out, devOut, n * sizeof(Complex), hipMemcpyDeviceToHost);

    // 释放显存
    hipFree(devOut);
    hipFree(devIn);
}

void dft(const Complex* in, size_t n, Complex* out, int forward=1)
{
    if (n <= 1024)
        dft1(in, n, out, forward);
    else
        dft2(in, n, out, forward);
}

void idft(const Complex* values, size_t n, Complex* coefficients)
{
    dft(values, n, coefficients, -1);
}

// 拆分系数向量
__global__
void split(const Complex* in, size_t n, Complex* out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    out[i] = in[i*2];
    out[i+n/2] = in[i*2+1];
}

// 循环拆分系数向量直到最底层
void splitAll(Complex** devData, Complex** devTemp, size_t n)
{
    // log(n)轮计算
    for (size_t groupSize = n; groupSize > 2; groupSize = groupSize/2)
    {
        size_t blocks = groupSize/2 <= 1024 ? 1 : ((groupSize/2 + 1023) / 1024);
        size_t threads = groupSize/2 <= 1024 ? (groupSize/2) : 1024;
        for (size_t i = 0; i < n; i+=groupSize)
        {
            split<<<blocks, threads>>>(*devData + i, groupSize, *devTemp + i);
        }
        cuda::std::swap(*devData, *devTemp);
    }
}

// 从底层分组向上合并
__global__ 
void combine(Complex* data, size_t n, size_t groupSize, Complex* temp, int forward=1)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i + groupSize/2 >= n)
        return;

    
    size_t groupIndex = i % groupSize;

    // 右半边和左半边一起计算出，因此右半直接跳过
    if (groupIndex >= groupSize/2)
        return;

    auto w = omega(forward*groupIndex, groupSize);
    temp[i] = data[i] + w * data[i + groupSize/2]; // 左半
    temp[i+ groupSize/2] = data[i] - w * data[i + groupSize/2]; // 右半
}


void fft(Complex* data, size_t n, int forward=1)
{
    Complex* temp = new Complex[n];

    // 分配显存
    Complex* devData;
    Complex* devTemp;
    hipMalloc(&devData, n * sizeof(Complex));
    hipMalloc(&devTemp, n * sizeof(Complex));

    hipMemcpy(devData, data, n * sizeof(Complex), hipMemcpyHostToDevice);

    // 将内存数据拷贝到显存
    splitAll(&devData, &devTemp, n);

    size_t blocks = n <= 1024 ? 1 : ((n + 1023) / 1024);
    size_t threads = n <= 1024 ? n : 1024;
    for (size_t groupSize = 2; groupSize <= n; groupSize *= 2)
    {
        combine<<<blocks, threads>>>(devData, n, groupSize, devTemp, forward);
        cuda::std::swap(devData, devTemp);
    }

    if (forward < 0)
    {
        div<<<blocks, threads>>>(devData, n, n);
    }

    // 将显存数据拷贝回内存
    hipMemcpy(data, devData, n * sizeof(Complex), hipMemcpyDeviceToHost);

    hipFree(devData);
    hipFree(devTemp);
}

void ifft(Complex* data, size_t n)
{
    fft(data, n, -1);
}

int main()
{
    // 分配内存并初始化数据
    Complex* data = new Complex[N];

    for (size_t i = 0; i < N; i++)
    {
        data[i].real(i);
        data[i].imag(i);
    }

    // 变换
    fft(data, N);
    for (size_t i = 0; i < N; i++)
    {
        printf("%f + %fi\n", data[i].real(), data[i].imag());
    }
    printf("\n");

    // 逆变换
    ifft(data, N);
    for (size_t i = 0; i < N; i++)
    {
        printf("%f + %fi\n", data[i].real(), data[i].imag());
    }

    delete[] data;

    return 0;
}